#include "../common/book.h"

int main( void ) {
    hipDeviceProp_t  prop;
    int dev;

    HANDLE_ERROR( hipGetDevice( &dev ) );
    printf( "ID of current CUDA device:  %d\n", dev );

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 3;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) ); // Returns the ID of the device that matches the properties specified in prop
    printf( "ID of CUDA device closest to revision 1.3:  %d\n", dev );

    HANDLE_ERROR( hipSetDevice( dev ) );
}
